#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<hip/hip_runtime.h>
#include<vector>
#include<math.h>
using namespace std;

void seq(vector<double>a, int n){
    double avg=0;
    for(int i=0;i<n;i++){
        avg+=a[i];
    }
    avg/=n;
    // cout<<avg<<"\n";
    double stdv=0;
    for(int i=0;i<n;i++){
        stdv+=(a[i]-avg)*(a[i]-avg);
    }
    stdv/=n;
    stdv=sqrt(stdv);
    cout<<stdv<<"\n";
}

__global__ void summ(double *d_in,double *d_out, int n){
    extern __shared__ double shared_mem[];
    int actind = blockIdx.x*blockDim.x+threadIdx.x;
    int ind = threadIdx.x;
    if(actind<n){
        shared_mem[ind] = d_in[actind];
    }else shared_mem[ind] = 0;
    __syncthreads();
    for(int steps=blockDim.x/2;steps>0;steps=steps/2){
        if(ind<steps){
            shared_mem[ind]+=shared_mem[ind+steps];
        }
        __syncthreads();
    }
    d_out[blockIdx.x] = shared_mem[0];
}

__global__ void subb(double *d_in, int n, double avg){
    int actind = blockIdx.x*blockDim.x+threadIdx.x;
    if(actind<n){
        d_in[actind] = (d_in[actind]-avg)*(d_in[actind]-avg);
    }
}

void parll(vector<double>a, int n){
    double *d_in, *d_out;
    double h_in[n];
    copy(a.begin(),a.end(),h_in);
    hipMalloc(&d_in,sizeof(double)*n);
    int blocks = 1+(n-1)/256;
    hipMalloc(&d_out,sizeof(double)*blocks);
    hipMemcpy(d_in,h_in,sizeof(double)*n,hipMemcpyHostToDevice);
    int sz=n;
    while(blocks>1){
        summ<<<blocks,256,sizeof(double)*256>>>(d_in,d_out,sz);
        hipFree(d_in);
        hipMalloc(&d_in,sizeof(double)*blocks);
        hipMemcpy(d_in,d_out,sizeof(double)*blocks,hipMemcpyDeviceToDevice);
        sz=blocks;
        blocks=1+(sz-1)/256;
        hipFree(d_out);
        hipMalloc(&d_out,sizeof(double)*blocks);
    }
    summ<<<1,256,sizeof(double)*256>>>(d_in,d_out,sz);
    double h_out[1];
    hipMemcpy(h_out,d_out,1*sizeof(double),hipMemcpyDeviceToHost);
    double avg=h_out[0]/n;
    hipFree(d_in);
    hipFree(d_out);
    hipMalloc(&d_in,sizeof(double)*n);
    hipMemcpy(d_in,h_in,sizeof(double)*n,hipMemcpyHostToDevice);
    subb<<<1+(n-1)/256,256>>>(d_in,n,avg);
    double fin[n];
    hipMemcpy(fin,d_in,sizeof(double)*n,hipMemcpyDeviceToHost);
    hipFree(&d_in);
    hipMalloc(&d_in,sizeof(double)*n);
    blocks = 1+(n-1)/256;
    hipMalloc(&d_out,sizeof(double)*blocks);
    hipMemcpy(d_in,fin,sizeof(double)*n,hipMemcpyHostToDevice);
    sz=n;
    while(blocks>1){
        summ<<<blocks,256,sizeof(double)*256>>>(d_in,d_out,sz);
        hipFree(d_in);
        hipMalloc(&d_in,sizeof(double)*blocks);
        hipMemcpy(d_in,d_out,sizeof(double)*blocks,hipMemcpyDeviceToDevice);
        sz=blocks;
        blocks=1+(sz-1)/256;
        hipFree(d_out);
        hipMalloc(&d_out,sizeof(double)*blocks);
    }
    summ<<<1,256,sizeof(double)*256>>>(d_in,d_out,sz);
    hipMemcpy(h_out,d_out,1*sizeof(double),hipMemcpyDeviceToHost);
    double ans=sqrt(h_out[0]/n);
    cout<<ans<<"\n";
}

int main(){
    srand(time(NULL));
    int n=100000;
    vector<double>a(n);
    for(int i=0;i<n;i++){
        a[i]=rand()%100;
    }
    seq(a,n);
    parll(a,n);
}
