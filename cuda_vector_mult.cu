#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
using namespace std;

void seq(int v1[], int v2[], int n, int m){
    int v3[m]={0};
    clock_t st = clock();
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            v3[i]+=v1[j]*v2[j*m+i];
        }
    }
    clock_t en = clock();
    cout<<"Sequential vector-matrix multiplication: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    int sum=0;
    for(int i=0;i<m;i++){
        sum+=v3[i];
    }
    cout<<sum<<"\n";
}

__global__ void multVector(int *d1_in, int *d2_in, int *d_out, int n, int m){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    if(ind<m){
        d_out[ind]=0;
        for(int i=0;i<n;i++){
            d_out[ind]+= d1_in[i]*d2_in[i*m+ind];
        }
    }
}

void parll(int v1[], int v2[], int n, int m){
    int *d1_in, *d2_in, *d_out;
    int h_out[m];
    hipMalloc(&d1_in,sizeof(int)*n);
    hipMalloc(&d2_in,sizeof(int)*n*m);
    hipMalloc(&d_out,sizeof(int)*m);
    hipMemcpy(d1_in,v1,sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(d2_in,v2,sizeof(int)*n*m,hipMemcpyHostToDevice);
    clock_t st = clock();
    multVector<<<1+(m-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d1_in,d2_in,d_out,n,m);
    clock_t en = clock();
    cout<<"Parallel vector-matrix multiplication: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    hipMemcpy(h_out, d_out, sizeof(int)*m,hipMemcpyDeviceToHost);
    int sum=0;
    for(int i=0;i<m;i++){
        sum+=h_out[i];
    }
    cout<<sum<<"\n"; //to confirm if the elements are the same
    hipFree(d1_in);
    hipFree(d2_in);
    hipFree(d_out);
}

int main(){
    srand(time(NULL));
    int n=1000;
    int m=1000;
    int v1[n];
    int v2[n*m];
    for(int i=0;i<n;i++){
        v1[i]=rand()%100;
    }
    for(int i=0;i<n*m;i++){
        v2[i]=rand()%100;
    }
    seq(v1,v2,n,m);
    parll(v1,v2,n,m);
}
