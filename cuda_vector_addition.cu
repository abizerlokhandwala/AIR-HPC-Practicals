#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
using namespace std;

void seq(int v1[], int v2[], int n, int m){
    int v3[n];
    clock_t st = clock();
    for(int i=0;i<n;i++){
        v3[i]=v1[i]+v2[i];
    }
    clock_t en = clock();
    cout<<"Sequential vector addition: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    int sum=0;
    for(int i=0;i<n;i++){
        sum+=v3[i];
    }
    cout<<sum<<"\n";
}

__global__ void addVector(int *d1_in, int *d2_in, int *d_out, int n){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    if(ind<n){
        d_out[ind] = d1_in[ind]+d2_in[ind];
    }
}

void parll(int v1[], int v2[], int n, int m){
    int *d1_in, *d2_in, *d_out;
    int h_out[n];
    hipMalloc(&d1_in,sizeof(int)*n);
    hipMalloc(&d2_in,sizeof(int)*n);
    hipMalloc(&d_out,sizeof(int)*n);
    hipMemcpy(d1_in,v1,sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(d2_in,v2,sizeof(int)*n,hipMemcpyHostToDevice);
    clock_t st = clock();
    addVector<<<1+(n-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d1_in,d2_in,d_out,n);
    clock_t en = clock();
    cout<<"Parallel vector addition: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    hipMemcpy(h_out, d_out, sizeof(int)*n,hipMemcpyDeviceToHost);
    int sum=0;
    for(int i=0;i<n;i++){
        sum+=h_out[i];
    }
    cout<<sum<<"\n"; //to confirm if the elements are the same
    hipFree(d1_in);
    hipFree(d2_in);
    hipFree(d_out);
}

int main(){
    srand(time(NULL));
    int n=500000;//10 fold speedup
    int m=n;
    int v1[n],v2[m];
    for(int i=0;i<n;i++){
        v1[i]=rand()%100;
        v2[i]=rand()%100;
    }
    seq(v1,v2,n,m);
    parll(v1,v2,n,m);
}
