#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256
using namespace std;

void seq(int v1[], int v2[], int n, int m, int k){
    int v3[n*k]={0};
    clock_t st = clock();
    for(int i=0;i<n;i++){
        for(int j=0;j<k;j++){
            for(int l=0;l<m;l++){
                v3[i*k+j] += v1[i*m+l]*v2[l*k+j];
            }
        }
    }
    clock_t en = clock();
    cout<<"Sequential matrix-matrix multiplication: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    int sum=0;
    for(int i=0;i<n*k;i++){
        sum+=v3[i];
        // cout<<v3[i]<<" ";
    }
    cout<<sum<<"\n";
}

__global__ void multMatrix(int *d1_in, int *d2_in, int *d_out, int n, int m, int k){
    int indx = threadIdx.x;
    int indy = threadIdx.y;
    int ind = indy*k+indx;
    //printf("%d %d\n",indy,indx);
    if(ind<n*k){
        d_out[ind] = 0;
        for(int i=0;i<m;i++){
            d_out[ind] += d1_in[indy*m+i]*d2_in[i*k+indx];
        }
    }
}

void parll(int v1[], int v2[], int n, int m, int k){
    int *d1_in, *d2_in, *d_out;
    int h_out[n*k];
    hipMalloc(&d1_in,sizeof(int)*n*m);
    hipMalloc(&d2_in,sizeof(int)*m*k);
    hipMalloc(&d_out,sizeof(int)*n*k);
    hipMemcpy(d1_in,v1,sizeof(int)*n*m,hipMemcpyHostToDevice);
    hipMemcpy(d2_in,v2,sizeof(int)*m*k,hipMemcpyHostToDevice);
    dim3 threads2D(k,n);
    dim3 blocks(1,1);
    clock_t st = clock();
    multMatrix<<<blocks,threads2D>>>(d1_in,d2_in,d_out,n,m,k);
    clock_t en = clock();
    cout<<"Parallel matrix-matrix multiplication: "<<double(en-st)/CLOCKS_PER_SEC<<"\n";
    hipMemcpy(h_out, d_out, sizeof(int)*n*k,hipMemcpyDeviceToHost);
    int sum=0;
    for(int i=0;i<n*k;i++){
        sum+=h_out[i];
        // cout<<h_out[i]<<" ";
    }
    cout<<sum<<"\n"; //to confirm if the elements are the same
    hipFree(d1_in);
    hipFree(d2_in);
    hipFree(d_out);
}

int main(){
    srand(time(NULL));
    int n=20;
    int m=50;
    int k=50;
    int v1[n*m];
    int v2[m*k];
    for(int i=0;i<n*m;i++){
        v1[i]=rand()%100;
    }
    for(int i=0;i<m*k;i++){
        v2[i]=rand()%100;
    }
    seq(v1,v2,n,m,k);
    parll(v1,v2,n,m,k);
}
